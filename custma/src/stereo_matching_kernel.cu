#include "hip/hip_runtime.h"
// Copyright 2022 Zhihao Liang
#include "stereo_matching.hpp"

/* NOTE: now ignore the crop_w and D */
const float EPSILON = 1e-8;

__device__ float query_ij(
    const float* __restrict__ val_ptr, // [H, W]
    const int32_t H,
    const int32_t W,
    const int32_t i,
    const int32_t j) {
    return (i < 0 || i >= H || j < 0 || j >= W) ? 0.f : val_ptr[i * W + j];
}

// KERNELS
__global__ void unfold_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t ks,
    const float* __restrict__ inputs_ptr, // [H, W]
    // output
    float* __restrict__ outputs_ptr // [H, W, ks * ks]
) {
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;
    const int32_t patch_i = h_idx + i - ks / 2;
    const int32_t patch_j = w_idx + j - ks / 2;
    const int32_t off = ks * ks;

    const float val = query_ij(inputs_ptr, H, W, patch_i, patch_j);
    outputs_ptr[h_idx * W * off + w_idx * off + i * ks + j] = val;
}

__global__ void get_ex2_exy_grad_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t ks,
    const float* __restrict__ cost_volume_grad_ptr, // [H, W - D, W]
    const float* __restrict__ ex2_ptr, // [H, W - D]
    const float* __restrict__ ey2_ptr, // [H, W]
    const float* __restrict__ exy_ptr, // [H, W - D, W]
    // output
    float* __restrict__ ex2_grad_ptr, // [H, W - D]
    float* __restrict__ exy_grad_ptr // [H, W - D, W]
) {
    // the coordinate of pixel
    const int32_t h_idx = threadIdx.x;
    const int32_t w_idx = blockIdx.x;
    const int32_t d_idx = blockIdx.y;

    const float ex2 = ex2_ptr[h_idx * (W - D) + w_idx];
    const float ey2 = ey2_ptr[h_idx * W + d_idx];
    const float exy = exy_ptr[h_idx * (W - D) * W + w_idx * W + d_idx];

    const float deno = 1 / (sqrtf(ex2 * ey2 + EPSILON)),
                deno3 = 1 / powf((sqrtf(ex2 * ey2 + EPSILON)), 3);
    const float cost_volume_grad =
        cost_volume_grad_ptr[h_idx * (W - D) * W + w_idx * W + d_idx];

    const float exy_grad = cost_volume_grad * deno;
    exy_grad_ptr[h_idx * (W - D) * W + w_idx * W + d_idx] = exy_grad;

    const float ex2_grad =
        -ey2_ptr[h_idx * W + d_idx] * (exy + EPSILON) * deno3 / 2;
    atomicAdd(&ex2_grad_ptr[h_idx * (W - D) + w_idx], ex2_grad);
}


__global__ void patch_grad_to_image_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t ks,
    const float* __restrict__ camera_patches_grad_ptr, // [H, W, ks, ks]
    // output
    float* __restrict__ camera_grad_ptr // [H, W]
) {
    // the coordinate of pixel
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    // relative coordinate in the patch
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;
    const int32_t off = ks * ks;

    const int32_t cam_i = h_idx + i - ks / 2;
    const int32_t cam_j = w_idx + j - ks / 2;
    if (cam_i < 0 || cam_i >= H || cam_j < 0 || cam_j >= W) {
        return;
    }
    atomicAdd(
        camera_grad_ptr + cam_i * W + cam_j,
        camera_patches_grad_ptr[h_idx * W * off + w_idx * off + i * ks + j]);
}


vector<Tensor> stereo::stereo_matching_forward(
    const Tensor& camera, // [H, W]
    const Tensor& projector, // [H, W]
    const int32_t D,
    const int32_t kernel_size) {
    // check
    CHECK_INPUT(camera);
    CHECK_INPUT(projector);

    // get parameters
    const int32_t H = camera.size(0), W = camera.size(1);
    const int32_t crop_w = W - D;
    assert(projector.size(0) == H && projector.size(1) == W);

    // unfold operation
    Tensor camera_patch = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor projector_patch = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    dim3 dim_grid(H, W);
    dim3 dim_block(kernel_size, kernel_size);
    unfold_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        camera.data_ptr<float>(),
        // output
        camera_patch.data_ptr<float>());
    unfold_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        projector.data_ptr<float>(),
        // output
        projector_patch.data_ptr<float>());
    Tensor camera_patch_mean = torch::mean(camera_patch, 2, true);
    Tensor projector_patch_mean = torch::mean(projector_patch, 2, true);
    camera_patch -= camera_patch_mean;
    projector_patch -= projector_patch_mean;

    Tensor ex2 =
        torch::bmm(
            camera_patch.reshape({H * W, 1, kernel_size * kernel_size}),
            camera_patch.reshape({H * W, kernel_size * kernel_size, 1}))
            .reshape({H, W});
    Tensor ey2 =
        torch::bmm(
            projector_patch.reshape({H * W, 1, kernel_size * kernel_size}),
            projector_patch.reshape({H * W, kernel_size * kernel_size, 1}))
            .reshape({H, W});

    Tensor exy = torch::bmm(camera_patch, projector_patch.permute({0, 2, 1}));
    Tensor cost_volume =
        (exy + EPSILON) / torch::sqrt(ex2 * ey2 + EPSILON).unsqueeze_(2);

    vector<Tensor> results(8);

    results[0] = ex2;
    results[1] = ey2;
    results[2] = exy;
    results[3] = camera_patch_mean;
    results[4] = projector_patch_mean;
    results[5] = cost_volume;
    results[6] = camera_patch;
    results[7] = projector_patch;

    return results;
}

vector<Tensor> stereo::stereo_matching_backward(
    const Tensor& cost_volume_grad, // [H, W - D, W]
    const Tensor& camera, // [H, W]
    const Tensor& projector, // [H, W]
    const Tensor& ex2,
    const Tensor& ey2,
    const Tensor& exy,
    const Tensor& ex2_mean,
    const Tensor& ey2_mean,
    const int32_t kernel_size,
    const bool record) {
    // check
    CHECK_INPUT(cost_volume_grad);
    CHECK_INPUT(camera);
    CHECK_INPUT(projector);
    CHECK_INPUT(ex2);
    CHECK_INPUT(ey2);
    CHECK_INPUT(exy);
    CHECK_INPUT(ex2_mean);
    CHECK_INPUT(ey2_mean);

    // get parameters
    const int32_t H = camera.size(0), W = camera.size(1),
                  crop_w = cost_volume_grad.size(1);
    const int32_t D = W - crop_w;

    Tensor ex2_grad = torch::zeros_like(ex2);
    Tensor exy_grad = torch::zeros_like(exy);
    Tensor camera_grad = torch::zeros_like(camera);
    Tensor camera_patch_grad = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    dim3 dim_grid_(crop_w, W);
    get_ex2_exy_grad_kernel<<<dim_grid_, H>>>(
        H,
        W,
        D,
        kernel_size,
        cost_volume_grad.data_ptr<float>(),
        ex2.data_ptr<float>(),
        ey2.data_ptr<float>(),
        exy.data_ptr<float>(),
        // output
        ex2_grad.data_ptr<float>(),
        exy_grad.data_ptr<float>());

    // unfold operation
    Tensor camera_patch = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor projector_patch = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    dim3 dim_grid(H, W);
    dim3 dim_block(kernel_size, kernel_size);
    unfold_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        camera.data_ptr<float>(),
        // output
        camera_patch.data_ptr<float>());
    unfold_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        projector.data_ptr<float>(),
        // output
        projector_patch.data_ptr<float>());
    Tensor camera_patch_mean = torch::mean(camera_patch, 2, true);
    Tensor projector_patch_mean = torch::mean(projector_patch, 2, true);
    camera_patch -= camera_patch_mean;
    projector_patch -= projector_patch_mean;

    camera_patch_grad +=
        torch::bmm(
            ex2_grad.reshape({H * W, 1, 1}),
            camera_patch.reshape({H * W, 1, kernel_size * kernel_size}))
            .reshape({H, W, kernel_size * kernel_size});
    camera_patch_grad += torch::bmm(exy_grad, projector_patch);

    dim3 img_dim_grid(H, W);
    patch_grad_to_image_kernel<<<img_dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        camera_patch_grad.data_ptr<float>(),
        // output
        camera_grad.data_ptr<float>());

    vector<Tensor> results(3);

    results[0] = camera_grad;
    results[1] = ex2_grad;
    results[2] = exy_grad;

    return results;
}
