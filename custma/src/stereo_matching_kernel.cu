#include "hip/hip_runtime.h"
// Copyright 2022 Zhihao Liang
#include "stereo_matching.hpp"

/* NOTE: now ignore the crop_w and D */
const float EPSILON = 1e-8;

__device__ float query_ij(
    const float* __restrict__ val_ptr, // [H, W]
    const int32_t H,
    const int32_t W,
    const int32_t i,
    const int32_t j) {
    return (i < 0 || i >= H || j < 0 || j >= W) ? 0.f : val_ptr[i * W + j];
}

// KERNELS
__global__ void self_cov_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t kernel_size,
    const float* __restrict__ inputs_ptr, // [H, W]
    // output
    float* __restrict__ mean_buffer, // [H, W - D]
    float* __restrict__ self_cov_ptr // [H, W - D]
) {
    // the coordinate of pixel
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    // relative coordinate in the patch
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;

    const int32_t patch_i = h_idx + i - kernel_size / 2;
    const int32_t patch_j = w_idx + D + j - kernel_size / 2;
    const float val = query_ij(inputs_ptr, H, W, patch_i, patch_j);

    atomicAdd(&mean_buffer[h_idx * (W - D) + w_idx], val);
    __syncthreads();

    const float _mean_tmp =
        mean_buffer[h_idx * (W - D) + w_idx] / (kernel_size * kernel_size);
    const float norm_val = val - _mean_tmp;
    atomicAdd(&self_cov_ptr[h_idx * (W - D) + w_idx], norm_val * norm_val);
    __syncthreads();
}

__global__ void cross_cov_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t kernel_size,
    const float* __restrict__ cam_ptr, // [H, W]
    const float* __restrict__ proj_ptr, // [H, W]
    const float* __restrict__ cam_mean_buffer, // [H, W - D]
    const float* __restrict__ proj_mean_buffer, // [H, W]
    // output
    float* __restrict__ cross_cov_ptr // [H, W - D, W]
) {
    // the coordinate of pixel
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    const int32_t d_idx = blockIdx.z;
    // relative coordinate in the patch
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;

    const float cam_mean = cam_mean_buffer[h_idx * (W - D) + w_idx];
    const float proj_mean = proj_mean_buffer[h_idx * W + d_idx];
    const int32_t cam_patch_i = h_idx + i - kernel_size / 2;
    const int32_t cam_patch_j = w_idx + D + j - kernel_size / 2;
    const float cam_val =
        query_ij(cam_ptr, H, W, cam_patch_i, cam_patch_j) - cam_mean;

    const int32_t proj_patch_i = h_idx + i - kernel_size / 2;
    const int32_t proj_patch_j = d_idx + j - kernel_size / 2;
    const float proj_val =
        query_ij(proj_ptr, H, W, proj_patch_i, proj_patch_j) - proj_mean;

    atomicAdd(
        &cross_cov_ptr[h_idx * (W - D) * W + w_idx * W + d_idx],
        cam_val * proj_val);
}

__global__ void cost_volume_kernel(
    const int32_t elements,
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const float* __restrict__ ex2_ptr, // [H, W - D]
    const float* __restrict__ ey2_ptr, // [H, W]
    const float* __restrict__ exy_ptr, // [H, W - D, W]
    // output
    float* __restrict__ cost_volume_ptr // [H, W - D, W]
) {
    const int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= elements) {
        return;
    }
    // the coordinate of pixel
    const int32_t d_idx = tid % W;
    const int32_t w_idx = (tid / W) % (W - D);
    const int32_t h_idx = tid / (W * (W - D));

    const float ex2 = ex2_ptr[h_idx * (W - D) + w_idx];
    const float ey2 = ey2_ptr[h_idx * W + d_idx];
    const float exy = exy_ptr[tid];

    cost_volume_ptr[tid] = (exy + EPSILON) / (sqrtf(ex2 * ey2 + EPSILON));
}

__global__ void get_self_grad_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t kernel_size,
    const float* __restrict__ inputs_ptr, // [H, W]
    const float* __restrict__ cov_avg_ptr, // [H, W - D]
    const float* __restrict__ cov_grad_ptr, // [H, W - D]
    const float* __restrict__ cov_ptr, // [H, W - D]
    // output
    float* __restrict__ input_grad_ptr // [H, W]
) {
    // the coordinate of pixel
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    // relative coordinate in the patch
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;

    const int32_t patch_i = h_idx + i - kernel_size / 2;
    const int32_t patch_j = w_idx + D + j - kernel_size / 2;

    if (patch_i < 0 || patch_i >= H || patch_j < 0 || patch_j >= W) {
        return;
    }

    const float val = query_ij(inputs_ptr, H, W, patch_i, patch_j);
    const float avg = cov_avg_ptr[h_idx * (W - D) + w_idx];
    const float grad = cov_grad_ptr[h_idx * (W - D) + w_idx];
    const float factor = 2 * (val - avg / (kernel_size * kernel_size));

    atomicAdd(&input_grad_ptr[patch_i * W + patch_j], factor * grad);
    __syncthreads();
}

__global__ void get_cross_grad_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t kernel_size,
    const bool record_grad,
    const float* __restrict__ camera_ptr, // [H, W]
    const float* __restrict__ projector_ptr, // [H, W]
    const float* __restrict__ cost_volume_grad_ptr, // [H, W - D, W]
    const float* __restrict__ ex2_ptr, // [H, W - D]
    const float* __restrict__ ey2_ptr, // [H, W]
    const float* __restrict__ exy_ptr, // [H, W - D, W]
    const float* __restrict__ ex2_mean_ptr, // [H, W - D]
    const float* __restrict__ ey2_mean_ptr, // [H, W]
    // output
    float* __restrict__ camera_grad_ptr, // [H, W]
    float* __restrict__ ex2_grad_ptr, // [H, W - D]
    float* __restrict__ exy_grad_ptr // [H, W - D, W]
) {
    // the coordinate of pixel
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    const int32_t d_idx = blockIdx.z;
    // relative coordinate in the patch
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;

    const float ex2 = ex2_ptr[h_idx * (W - D) + w_idx];
    const float ey2 = ey2_ptr[h_idx * W + d_idx];
    const float exy = exy_ptr[h_idx * (W - D) * W + w_idx * W + d_idx];

    const float cost_volume_grad =
        cost_volume_grad_ptr[h_idx * (W - D) * W + w_idx * W + d_idx];

    const float factor = (sqrtf(ex2 * ey2 + EPSILON));
    const int32_t cam_patch_i = h_idx + i - kernel_size / 2;
    const int32_t cam_patch_j = w_idx + D + j - kernel_size / 2;
    const int32_t proj_patch_i = h_idx + i - kernel_size / 2;
    const int32_t proj_patch_j = d_idx + j - kernel_size / 2;

    // record the intermediate gradient for debug
    const float ex2_grad =
        -ey2_ptr[h_idx * W + d_idx] * (exy + EPSILON) / (2 * powf(factor, 3));
    const float exy_grad = cost_volume_grad / factor;
    if (record_grad) {
        if (i == 0 && j == 0) {
            atomicAdd(&ex2_grad_ptr[h_idx * (W - D) + w_idx], ex2_grad);
            exy_grad_ptr[h_idx * (W - D) * W + w_idx * W + d_idx] = exy_grad;
        }
    }

    // skip while outside the image
    if (cam_patch_i < 0 || cam_patch_i >= H || cam_patch_j < 0 ||
        cam_patch_j >= W) {
        return;
    }

    /* ex2 term */
    __syncthreads();
    const float cam_val = query_ij(camera_ptr, H, W, cam_patch_i, cam_patch_j);
    const float cam_grad_ex2_term =
        2 * (cam_val - ex2_mean_ptr[h_idx * (W - D) + w_idx]) * ex2_grad;
    atomicAdd(
        &camera_grad_ptr[cam_patch_i * W + cam_patch_j], cam_grad_ex2_term);
    __syncthreads();

    /* exy term */
    const float proj_val =
        query_ij(projector_ptr, H, W, proj_patch_i, proj_patch_j);
    const float cam_grad_exy_term =
        (proj_val - ey2_mean_ptr[h_idx * W + d_idx]) * exy_grad;

    atomicAdd(
        &camera_grad_ptr[cam_patch_i * W + cam_patch_j], cam_grad_exy_term);
}

__global__ void exy_grad_to_image_kernel2(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t kernel_size,
    const float* __restrict__ exy_grad_ptr, // [H, W - D, W]
    const float* __restrict__ projector_ptr, // [H, W]
    const float* __restrict__ ey2_mean_ptr, // [H, W]
    // output
    float* __restrict__ camera_grad_ptr // [H, W]
) {
    // the coordinate of pixel
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    const int32_t d_idx = blockIdx.z;
    // relative coordinate in the patch
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;

    const float exy_grad =
        exy_grad_ptr[h_idx * (W - D) * W + w_idx * W + d_idx];

    const int32_t cam_patch_i = h_idx + i - kernel_size / 2;
    const int32_t cam_patch_j = w_idx + D + j - kernel_size / 2;
    const int32_t proj_patch_i = h_idx + i - kernel_size / 2;
    const int32_t proj_patch_j = d_idx + j - kernel_size / 2;
    const float proj_val =
        query_ij(projector_ptr, H, W, proj_patch_i, proj_patch_j);
    const float grad = (proj_val - ey2_mean_ptr[h_idx * W + d_idx]) * exy_grad;

    if (cam_patch_i < 0 || cam_patch_i >= H || cam_patch_j < 0 ||
        cam_patch_j >= W) {
        return;
    }
    atomicAdd(&camera_grad_ptr[cam_patch_i * W + cam_patch_j], grad);
    __syncthreads();
}

vector<Tensor> stereo::stereo_matching_forward(
    const Tensor& camera, // [H, W]
    const Tensor& projector, // [H, W]
    const int32_t D,
    const int32_t kernel_size) {
    // check
    CHECK_INPUT(camera);
    CHECK_INPUT(projector);

    // get parameters
    const int32_t H = camera.size(0), W = camera.size(1);
    const int32_t crop_w = W - D;
    assert(projector.size(0) == H && projector.size(1) == W);

    /* self cov */
    Tensor ex2_mean = torch::zeros(
        {H, crop_w},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor ex2 = torch::zeros(
        {H, crop_w},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    dim3 dim_block(kernel_size, kernel_size);
    dim3 ex2_dim_grid(H, crop_w);
    self_cov_kernel<<<ex2_dim_grid, dim_block>>>(
        H,
        W,
        D,
        kernel_size,
        camera.data_ptr<float>(),
        // output
        ex2_mean.data_ptr<float>(),
        ex2.data_ptr<float>());
    ex2_mean /= (kernel_size * kernel_size);

    Tensor ey2_mean = torch::zeros(
        {H, W},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor ey2 = torch::zeros(
        {H, W},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    dim3 ey2_dim_grid(H, W);
    self_cov_kernel<<<ey2_dim_grid, dim_block>>>(
        H,
        W,
        0,
        kernel_size,
        projector.data_ptr<float>(),
        // output
        ey2_mean.data_ptr<float>(),
        ey2.data_ptr<float>());
    ey2_mean /= (kernel_size * kernel_size);

    /* cross cov */
    Tensor exy = torch::zeros(
        {H, crop_w, W},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    // NOTE: atomicAdd is too slow for sync while the blocks are too many
    dim3 exy_dim_grid(H, crop_w, W);
    cross_cov_kernel<<<exy_dim_grid, dim_block>>>(
        H,
        W,
        0,
        kernel_size,
        camera.data_ptr<float>(),
        projector.data_ptr<float>(),
        ex2_mean.data_ptr<float>(),
        ey2_mean.data_ptr<float>(),
        // output
        exy.data_ptr<float>());

    Tensor cost_volume = torch::zeros(
        {H, crop_w, W},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    const int32_t elements = H * crop_w * W, threads = 1024;
    const int32_t blocks = ceil((elements - 1) / threads) + 1;
    cost_volume_kernel<<<blocks, threads>>>(
        elements,
        H,
        W,
        D,
        ex2.data_ptr<float>(),
        ey2.data_ptr<float>(),
        exy.data_ptr<float>(),
        // output
        cost_volume.data_ptr<float>());

    vector<Tensor> results(6);

    results[0] = ex2;
    results[1] = ey2;
    results[2] = exy;
    results[3] = ex2_mean;
    results[4] = ey2_mean;
    results[5] = cost_volume;

    return results;
}

vector<Tensor> stereo::stereo_matching_backward(
    const Tensor& cost_volume_grad, // [H, W - D, W]
    const Tensor& camera, // [H, W]
    const Tensor& projector, // [H, W]
    const Tensor& ex2,
    const Tensor& ey2,
    const Tensor& exy,
    const Tensor& ex2_mean,
    const Tensor& ey2_mean,
    const int32_t kernel_size,
    const bool record) {
    // check
    CHECK_INPUT(cost_volume_grad);
    CHECK_INPUT(camera);
    CHECK_INPUT(projector);
    CHECK_INPUT(ex2);
    CHECK_INPUT(ey2);
    CHECK_INPUT(exy);
    CHECK_INPUT(ex2_mean);
    CHECK_INPUT(ey2_mean);

    // get parameters
    const int32_t H = camera.size(0), W = camera.size(1),
                  crop_w = cost_volume_grad.size(1);
    const int32_t D = W - crop_w;

    Tensor ex2_grad;
    Tensor exy_grad;
    if (record) {
        ex2_grad = torch::zeros_like(ex2);
        exy_grad = torch::zeros_like(exy);
    }
    Tensor camera_grad = torch::zeros_like(camera);

    dim3 dim_block(kernel_size, kernel_size);
    dim3 dim_grid(H, crop_w, W);
    get_cross_grad_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        D,
        kernel_size,
        record,
        camera.data_ptr<float>(),
        projector.data_ptr<float>(),
        cost_volume_grad.data_ptr<float>(),
        ex2.data_ptr<float>(),
        ey2.data_ptr<float>(),
        exy.data_ptr<float>(),
        ex2_mean.data_ptr<float>(),
        ey2_mean.data_ptr<float>(),
        // output
        camera_grad.data_ptr<float>(),
        record ? ex2_grad.data_ptr<float>() : nullptr,
        record ? exy_grad.data_ptr<float>() : nullptr);

    vector<Tensor> results(3);

    results[0] = camera_grad;
    results[1] = ex2_grad;
    results[2] = exy_grad;

    return results;
}
