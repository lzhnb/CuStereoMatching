#include "hip/hip_runtime.h"
// Copyright 2022 Zhihao Liang
#include "stereo_matching.hpp"

#define MAX_KERNEL_SIZE 15

__device__ const float EPSILON = 1e-8;

__device__ float query_ij(
    const float* __restrict__ img_ptr, // [H, W]
    const int32_t H,
    const int32_t W,
    const int32_t i,
    const int32_t j) {
    return (i < 0 || i >= H || j < 0 || j >= W) ? 0.f : img_ptr[i * W + j];
}

/* NOTE: now ignore the crop_w and D */

// KERNELS
__global__ void unfold_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t ks,
    const float* __restrict__ inputs_ptr, // [H, W]
    // output
    float* __restrict__ outputs_ptr // [H, W, ks * ks]
) {
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;
    const int32_t patch_i = h_idx + i - ks / 2;
    const int32_t patch_j = w_idx + j - ks / 2;
    const int32_t off = ks * ks;

    const float val = query_ij(inputs_ptr, H, W, patch_i, patch_j);
    outputs_ptr[h_idx * W * off + w_idx * off + i * ks + j] = val;
}

__global__ void gather_exy_cost_volume_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const float* __restrict__ full_exy_ptr, // [H, W, W]
    const float* __restrict__ full_cost_volume_ptr, // [H, W, W]
    // output
    float* __restrict__ exy_ptr, // [H, W, D]
    float* __restrict__ cost_volume_ptr // [H, W, D]
) {
    const int32_t h_idx = blockIdx.x, w_idx = blockIdx.y, d_idx = threadIdx.x;

    exy_ptr[h_idx * W * D + (w_idx + D) * D + d_idx] =
        full_exy_ptr[h_idx * W * W + (w_idx + D) * W + w_idx + d_idx];
    cost_volume_ptr[h_idx * W * D + (w_idx + D) * D + d_idx] =
        full_cost_volume_ptr[h_idx * W * W + (w_idx + D) * W + w_idx + d_idx];
}

__global__ void get_patches_grad_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t ks,
    const float* __restrict__ cost_volume_grad, // [H, W, D]
    const float* __restrict__ cam_ptr, // [H, W]
    const float* __restrict__ proj_ptr, // [H, W]
    const float* __restrict__ ex2_ptr, // [H, W]
    const float* __restrict__ ey2_ptr, // [H, W]
    const float* __restrict__ exy_ptr, // [H, W, D]
    const float* __restrict__ cam_patch_mean_ptr, // [H, W]
    const float* __restrict__ proj_patch_mean_ptr, // [H, W]
    // output
    float* __restrict__ camera_patches_grad_ptr // [H, W, ks, ks]
) {
    const int32_t h_idx = blockIdx.x, w_idx = blockIdx.y;
    const int32_t row = threadIdx.x, col = threadIdx.y;

    const float cam_mean = cam_patch_mean_ptr[h_idx * W + w_idx + D];

    const int32_t cam_i = h_idx + row - ks / 2;
    const int32_t cam_j = w_idx + col - ks / 2 + D;
    const float cam = query_ij(cam_ptr, H, W, cam_i, cam_j) - cam_mean;

    const float ex2 = ex2_ptr[h_idx * W + w_idx + D];

    for (int32_t d_idx = 0; d_idx < D; ++d_idx) {
        const float proj_mean = proj_patch_mean_ptr[h_idx * W + w_idx + d_idx];

        const int32_t proj_i = h_idx + row - ks / 2;
        const int32_t proj_j = d_idx + col - ks / 2 + w_idx;
        const float proj = query_ij(proj_ptr, H, W, proj_i, proj_j) - proj_mean;

        const float ey2 = ey2_ptr[h_idx * W + w_idx + d_idx];
        const float exy = exy_ptr[h_idx * W * D + (w_idx + D) * D + d_idx];

        const float cost_grad =
            cost_volume_grad[h_idx * W * D + (w_idx + D) * D + d_idx];
        const int32_t off_w = ks * ks, off_h = ks * ks * W;
        float* curr_camera_patches_grad_ptr =
            camera_patches_grad_ptr + h_idx * off_h + (w_idx + D) * off_w;

        // calculate 1 time to save time
        const float deno = 1 / (sqrtf(ex2 * ey2 + EPSILON)),
                    deno3 = 1 / powf((sqrtf(ex2 * ey2 + EPSILON)), 3);

        // exy term
        float exy_factor = proj * deno;
        // ex2 term
        float ex2_factor = -(ey2 * cam * (exy + EPSILON)) * deno3;
        const float grad = cost_grad * (exy_factor + ex2_factor);
        curr_camera_patches_grad_ptr[row * ks + col] += grad;
    }
}

__global__ void patches_grad_to_image_kernel(
    const int32_t elements,
    const int32_t H,
    const int32_t W,
    const int32_t ks,
    const float* __restrict__ camera_patches_grad_ptr, // [H, W, ks, ks]
    // output
    float* __restrict__ camera_grad_ptr // [H, W]
) {
    const int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= elements) {
        return;
    }
    const int32_t k2_idx = tid % ks;
    const int32_t k1_idx = (tid / ks) % ks;
    const int32_t w_idx = (tid / (ks * ks)) % W;
    const int32_t h_idx = tid / (W * ks * ks);

    const int32_t cam_i = h_idx + k1_idx - ks / 2;
    const int32_t cam_j = w_idx + k2_idx - ks / 2;
    if (cam_i < 0 || cam_i >= H || cam_j < 0 || cam_j >= W) {
        return;
    }
    atomicAdd(
        camera_grad_ptr + cam_i * W + cam_j, camera_patches_grad_ptr[tid]);
}

vector<Tensor> stereo::stereo_matching_forward(
    const Tensor& camera, // [H, W]
    const Tensor& projector, // [H, W]
    const int32_t D,
    const int32_t kernel_size) {
    // check
    CHECK_INPUT(camera);
    CHECK_INPUT(projector);

    // get parameters
    const int32_t H = camera.size(0), W = camera.size(1);
    const int32_t crop_w = W - D;
    assert(projector.size(0) == H && projector.size(1) == W);

    // unfold operation
    Tensor camera_patch = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor projector_patch = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    dim3 dim_grid(H, W);
    dim3 dim_block(kernel_size, kernel_size);
    unfold_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        camera.data_ptr<float>(),
        // output
        camera_patch.data_ptr<float>());
    unfold_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        projector.data_ptr<float>(),
        // output
        projector_patch.data_ptr<float>());
    Tensor camera_patch_mean = torch::mean(camera_patch, 2, true);
    Tensor projector_patch_mean = torch::mean(projector_patch, 2, true);
    camera_patch -= camera_patch_mean;
    projector_patch -= projector_patch_mean;

    Tensor ex2 =
        torch::bmm(
            camera_patch.reshape({H * W, 1, kernel_size * kernel_size}),
            camera_patch.reshape({H * W, kernel_size * kernel_size, 1}))
            .reshape({H, W, 1});
    Tensor ey2 =
        torch::bmm(
            projector_patch.reshape({H * W, 1, kernel_size * kernel_size}),
            projector_patch.reshape({H * W, kernel_size * kernel_size, 1}))
            .reshape({H, -1, W});

    Tensor full_exy =
        torch::bmm(camera_patch, projector_patch.permute({0, 2, 1}));
    Tensor full_cost_volume =
        (full_exy + EPSILON) / torch::sqrt(torch::bmm(ex2, ey2) + EPSILON);

    Tensor exy = torch::zeros(
        {H, W, D},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor cost_volume = torch::zeros_like(exy);
    dim3 block_dim(H, crop_w);
    gather_exy_cost_volume_kernel<<<block_dim, D>>>(
        H,
        W,
        D,
        full_exy.data_ptr<float>(),
        full_cost_volume.data_ptr<float>(),
        // output
        exy.data_ptr<float>(),
        cost_volume.data_ptr<float>());

    vector<Tensor> results(6);

    results[0] = ex2;
    results[1] = ey2;
    results[2] = exy;
    results[3] = cost_volume;
    results[4] = camera_patch_mean;
    results[5] = projector_patch_mean;

    return results;
}

Tensor stereo::stereo_matching_backward(
    const Tensor& cost_volume_grad, // [H, W, D]
    const Tensor& camera, // [H, W]
    const Tensor& projector, // [H, W]
    const Tensor& ex2, // [H, W]
    const Tensor& ey2, // [H, W]
    const Tensor& exy, // [H, W, D]
    const Tensor& cam_patch_mean, // [H, W]
    const Tensor& proj_patch_mean, // [H, W]
    const int32_t kernel_size) {
    // check
    CHECK_INPUT(cost_volume_grad);

    // get parameters
    const int32_t H = cost_volume_grad.size(0), W = cost_volume_grad.size(1),
                  D = cost_volume_grad.size(2);
    const int32_t crop_w = W - D;

    assert(kernel_size <= MAX_KERNEL_SIZE);
    Tensor camera_patches_grad = torch::zeros(
        {H, W, kernel_size, kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    const dim3 dim_block(H, crop_w), thread_block(kernel_size, kernel_size);
    get_patches_grad_kernel<<<dim_block, thread_block>>>(
        H,
        W,
        D,
        kernel_size,
        cost_volume_grad.data_ptr<float>(),
        camera.data_ptr<float>(),
        projector.data_ptr<float>(),
        ex2.data_ptr<float>(),
        ey2.data_ptr<float>(),
        exy.data_ptr<float>(),
        cam_patch_mean.data_ptr<float>(),
        proj_patch_mean.data_ptr<float>(),
        // output
        camera_patches_grad.data_ptr<float>());

    Tensor camera_grad = torch::zeros(
        {H, W},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    const int32_t elements2 = H * W * kernel_size * kernel_size, threads = 1024;
    const int32_t blocks2 = ceil((elements2 - 1) / threads) + 1;
    patches_grad_to_image_kernel<<<blocks2, threads>>>(
        elements2,
        H,
        W,
        kernel_size,
        camera_patches_grad.data_ptr<float>(),
        // output
        camera_grad.data_ptr<float>());

    return camera_grad;
}
