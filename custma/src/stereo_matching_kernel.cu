#include "hip/hip_runtime.h"
// Copyright 2022 Zhihao Liang
#include "stereo_matching.hpp"

#define MAX_KERNEL_SIZE 15

__device__ const float EPSILON = 1e-8;

__device__ float query_ij(
    const float* __restrict__ img_ptr, // [H, W]
    const int32_t H,
    const int32_t W,
    const int32_t i,
    const int32_t j) {
    return (i < 0 || i >= H || j < 0 || j >= W) ? 0.f : img_ptr[i * W + j];
}

/* NOTE: now ignore the crop_w and D */

// KERNELS
__global__ void unfold_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t ks,
    const float* __restrict__ inputs_ptr, // [H, W]
    // output
    float* __restrict__ outputs_ptr // [H, W, ks * ks]
) {
    const int32_t h_idx = blockIdx.x;
    const int32_t w_idx = blockIdx.y;
    const int32_t i = threadIdx.x;
    const int32_t j = threadIdx.y;
    const int32_t patch_i = h_idx + i - ks / 2;
    const int32_t patch_j = w_idx + j - ks / 2;
    const int32_t off = ks * ks;

    const float val = query_ij(inputs_ptr, H, W, patch_i, patch_j);
    outputs_ptr[h_idx * W * off + w_idx * off + i * ks + j] = val;
}

__global__ void forward_cost_volume_kernel(
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t kernel_size,
    const float* __restrict__ cam_ptr, // [H, W]
    const float* __restrict__ proj_ptr, // [H, W]
    // output
    float* __restrict__ cost_volume_ptr // [H, crop_w, D + 1]
) {
    const int32_t h_idx = blockIdx.x, w_idx = blockIdx.y, d_idx = threadIdx.x;

    float cam_patch[MAX_KERNEL_SIZE][MAX_KERNEL_SIZE];
    float proj_patch[MAX_KERNEL_SIZE][MAX_KERNEL_SIZE];

    // loop patch to get the mean value
    float cam_mean = 0, proj_mean = 0;
#pragma unroll
    for (int32_t row = 0; row < kernel_size; ++row) {
        for (int32_t col = 0; col < kernel_size; ++col) {
            const int32_t cam_i = h_idx + row - kernel_size / 2,
                          cam_j = w_idx + col - kernel_size / 2,
                          proj_i = h_idx + row - kernel_size / 2,
                          proj_j = d_idx + col - kernel_size / 2;
            const float cam = query_ij(cam_ptr, H, W, cam_i, cam_j);
            const float proj = query_ij(proj_ptr, H, W, proj_i, proj_j);
            cam_patch[row][col] = cam;
            proj_patch[row][col] = proj;
            cam_mean += cam;
            proj_mean += proj;
        }
    }
    cam_mean /= (kernel_size * kernel_size);
    proj_mean /= (kernel_size * kernel_size);

    float exy = 0, ex2 = 0, ey2 = 0;
#pragma unroll
    for (int32_t row = 0; row < kernel_size; ++row) {
        for (int32_t col = 0; col < kernel_size; ++col) {
            const float cam = cam_patch[row][col] - cam_mean;
            const float proj = proj_patch[row][col] - proj_mean;

            exy += cam * proj;
            ex2 += cam * cam;
            ey2 += proj * proj;
        }
    }
    cost_volume_ptr[h_idx * (W * (W - D)) + w_idx * W + d_idx] =
        (exy + EPSILON) / (sqrtf(ex2 * ey2 + EPSILON));
}

__global__ void get_patches_grad_kernel(
    // const int32_t elements,
    const int32_t H,
    const int32_t W,
    const int32_t D,
    const int32_t ks,
    const float* __restrict__ cost_volume_grad, // [H, crop_w, D + 1]
    const float* __restrict__ cam_ptr, // [H, W]
    const float* __restrict__ proj_ptr, // [H, W]
    const float* __restrict__ ex2_ptr, // [H, W]
    const float* __restrict__ ey2_ptr, // [H, W]
    const float* __restrict__ exy_ptr, // [H, W, W]
    const float* __restrict__ cam_patch_mean_ptr, // [H, W]
    const float* __restrict__ proj_patch_mean_ptr, // [H, W]
    // output
    float* __restrict__ camera_patches_grad_ptr // [H, W, ks, ks]
) {
    const int32_t h_idx = blockIdx.x, w_idx = blockIdx.y;
    const int32_t row = threadIdx.x, col = threadIdx.y;

    const float cam_mean = cam_patch_mean_ptr[h_idx * W + w_idx];

    const int32_t cam_i = h_idx + row - ks / 2;
    const int32_t cam_j = w_idx + col - ks / 2;
    const float cam = query_ij(cam_ptr, H, W, cam_i, cam_j) - cam_mean;

    const float ex2 = ex2_ptr[h_idx * W + w_idx];

    for (int32_t d_idx = 0; d_idx < W; ++d_idx) {
        const float proj_mean = proj_patch_mean_ptr[h_idx * W + d_idx];

        const int32_t proj_i = h_idx + row - ks / 2;
        const int32_t proj_j = d_idx + col - ks / 2;
        const float proj = query_ij(proj_ptr, H, W, proj_i, proj_j) - proj_mean;

        const float ey2 = ey2_ptr[h_idx * W + d_idx];
        const float exy = exy_ptr[h_idx * W * W + w_idx * W + d_idx];

        const float cost_grad =
            cost_volume_grad[h_idx * W * W + w_idx * W + d_idx];
        const int32_t off_w = ks * ks, off_h = ks * ks * W;
        float* curr_camera_patches_grad_ptr =
            camera_patches_grad_ptr + h_idx * off_h + w_idx * off_w;

        // calculate 1 time to save time
        const float deno = 1 / (sqrtf(ex2 * ey2 + EPSILON)),
                    deno3 = 1 / powf((sqrtf(ex2 * ey2 + EPSILON)), 3);

        // exy term
        float exy_factor = proj * deno;
        // ex2 term
        float ex2_factor = -(ey2 * cam * (exy + EPSILON)) * deno3;
        const float grad = cost_grad * (exy_factor + ex2_factor);
        curr_camera_patches_grad_ptr[row * ks + col] += grad;
    }
}

__global__ void patches_grad_to_image_kernel(
    const int32_t elements,
    const int32_t H,
    const int32_t W,
    const int32_t ks,
    const float* __restrict__ camera_patches_grad_ptr, // [H, W, ks, ks]
    // output
    float* __restrict__ camera_grad_ptr // [H, W]
) {
    const int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= elements) {
        return;
    }
    // const int32_t crop_w = W - D;
    // const int32_t d_idx = tid % (D + 1);
    // const int32_t w_idx = (tid / (D + 1)) % crop_w;
    // const int32_t h_idx = tid / ((D + 1) * crop_w);
    const int32_t k2_idx = tid % ks;
    const int32_t k1_idx = (tid / ks) % ks;
    const int32_t w_idx = (tid / (ks * ks)) % W;
    const int32_t h_idx = tid / (W * ks * ks);

    const int32_t cam_i = h_idx + k1_idx - ks / 2;
    const int32_t cam_j = w_idx + k2_idx - ks / 2;
    if (cam_i < 0 || cam_i >= H || cam_j < 0 || cam_j >= W) {
        return;
    }
    atomicAdd(
        camera_grad_ptr + cam_i * W + cam_j, camera_patches_grad_ptr[tid]);
}

vector<Tensor> stereo::stereo_matching_forward(
    const Tensor& camera, // [H, W]
    const Tensor& projector, // [H, W]
    const int32_t D,
    const int32_t kernel_size) {
    // check
    CHECK_INPUT(camera);
    CHECK_INPUT(projector);

    // get parameters
    const int32_t H = camera.size(0), W = camera.size(1);
    const int32_t crop_w = W - D;
    assert(projector.size(0) == H && projector.size(1) == W);

    // unfold operation
    Tensor camera_patch = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    Tensor projector_patch = torch::zeros(
        {H, W, kernel_size * kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    dim3 dim_grid(H, W);
    dim3 dim_block(kernel_size, kernel_size);
    unfold_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        camera.data_ptr<float>(),
        // output
        camera_patch.data_ptr<float>());
    unfold_kernel<<<dim_grid, dim_block>>>(
        H,
        W,
        kernel_size,
        projector.data_ptr<float>(),
        // output
        projector_patch.data_ptr<float>());
    Tensor camera_patch_mean = torch::mean(camera_patch, 2, true);
    Tensor projector_patch_mean = torch::mean(projector_patch, 2, true);
    camera_patch -= camera_patch_mean;
    projector_patch -= projector_patch_mean;

    Tensor ex2 =
        torch::bmm(
            camera_patch.reshape({H * W, 1, kernel_size * kernel_size}),
            camera_patch.reshape({H * W, kernel_size * kernel_size, 1}))
            .reshape({H, W, 1});
    Tensor ey2 =
        torch::bmm(
            projector_patch.reshape({H * W, 1, kernel_size * kernel_size}),
            projector_patch.reshape({H * W, kernel_size * kernel_size, 1}))
            .reshape({H, -1, W});

    Tensor exy = torch::bmm(camera_patch, projector_patch.permute({0, 2, 1}));

    Tensor cost_volume =
        (exy + EPSILON) / torch::sqrt(torch::bmm(ex2, ey2) + EPSILON);

    vector<Tensor> results(6);

    results[0] = ex2;
    results[1] = ey2;
    results[2] = exy;
    results[3] = cost_volume;
    results[4] = camera_patch_mean;
    results[5] = projector_patch_mean;

    return results;
}

Tensor stereo::stereo_matching_backward(
    const Tensor& cost_volume_grad, // [H, crop_w, D + 1]
    const Tensor& camera, // [H, W]
    const Tensor& projector, // [H, W]
    const Tensor& ex2, // [H, W]
    const Tensor& ey2, // [H, W]
    const Tensor& exy, // [H, W, W]
    const Tensor& cam_patch_mean, // [H, W]
    const Tensor& proj_patch_mean, // [H, W]
    const int32_t kernel_size) {
    // check
    CHECK_INPUT(cost_volume_grad);

    // get parameters
    const int32_t H = cost_volume_grad.size(0), W = cost_volume_grad.size(1),
                  D = cost_volume_grad.size(2);
    // const int32_t H = cost_volume_grad.size(0), crop_w =
    // cost_volume_grad.size(1), D = cost_volume_grad.size(2) - 1; const int32_t
    // W = crop_w + D;
    const int32_t elements1 = H * W, threads = 1024;
    // const int32_t elements1 = H * crop_w * (D + 1), threads = 1024;

    assert(kernel_size <= MAX_KERNEL_SIZE);
    Tensor camera_patches_grad = torch::zeros(
        {H, W, kernel_size, kernel_size},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));

    const int32_t blocks1 = ceil((elements1 - 1) / threads) + 1;
    const dim3 dim_block(H, W), thread_block(kernel_size, kernel_size);
    get_patches_grad_kernel<<<dim_block, thread_block>>>(
        // elements1,
        H,
        W,
        D,
        kernel_size,
        cost_volume_grad.data_ptr<float>(),
        camera.data_ptr<float>(),
        projector.data_ptr<float>(),
        ex2.data_ptr<float>(),
        ey2.data_ptr<float>(),
        exy.data_ptr<float>(),
        cam_patch_mean.data_ptr<float>(),
        proj_patch_mean.data_ptr<float>(),
        // output
        camera_patches_grad.data_ptr<float>());

    Tensor camera_grad = torch::zeros(
        {H, W},
        torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA));
    const int32_t elements2 = H * W * kernel_size * kernel_size;
    const int32_t blocks2 = ceil((elements2 - 1) / threads) + 1;
    patches_grad_to_image_kernel<<<blocks2, threads>>>(
        elements2,
        H,
        W,
        kernel_size,
        camera_patches_grad.data_ptr<float>(),
        // output
        camera_grad.data_ptr<float>());

    return camera_grad;
}
